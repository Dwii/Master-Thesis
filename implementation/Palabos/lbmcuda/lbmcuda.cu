#include "hip/hip_runtime.h"
/*!
 * \file    lbmcuda.c
 * \brief   LBM cuda library
 * \author  Adrien Python
 * \version 1.0
 * \date    14.06.2017
 */

#include "lbmcuda.h"

#include <stdio.h>

#define BLOCK_SIZE 64

#define SQUARE(a) ((a)*(a))
#define GPU_SQUARE(a) __dmul_rn(a,a)

#define IDX(x, y, z, nx, ny, nz) ((x+(nx))%(nx) + ((y+(ny))%(ny) + ( (z+(nz))%(nz) )*(ny))*(nx) )

typedef struct {
    bool* obstacles;
    lbm_lattices f0;
    lbm_lattices f1;
} lbm_vars;

struct lbm_simulation {
    lbm_vars d_vars;
    dim3 dimComputationGrid, dimComputationBlock;
    dim3 dimRightWallGrid, dimRightWallBlock;
    size_t shared_mem_size;
    bool switch_f0_f1;
    size_t nx, ny, nz;
    double omega;
};

#define HANDLE_ERROR(ans) (handleError((ans), __FILE__, __LINE__))
inline void handleError(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(EXIT_FAILURE);
   }
}

#define HANDLE_KERNEL_ERROR(...) \
do {                                         \
    __VA_ARGS__;                             \
/*    HANDLE_ERROR( hipPeekAtLastError() );  */ \
/*    HANDLE_ERROR( hipDeviceSynchronize() );*/ \
} while(0)


#define EQUILIBRIUM(rho, t, cu, usqr) __dmul_rn(__dmul_rn(rho, (t)), __dadd_rn(__dadd_rn(__dadd_rn(1, cu) , __dmul_rn(0.5, GPU_SQUARE(cu))), - usqr) )

__device__ static void equilibrium(double* ne, double* e, double* se, double* n, double* c, double* s, double* nw, double* w, double* sw, 
                                     double* te, double* tn, double* tc, double* ts, double* tw,
                                     double* be, double* bn, double* bc, double* bs, double* bw,
                                     double rho, double u0, double u1, double u2)
{
    double usqr = __dmul_rn(3./2, __dadd_rn( __dadd_rn( GPU_SQUARE(u0), GPU_SQUARE(u1)), GPU_SQUARE(u2) ));

    { double cu = 3 * (  u0 +  u1 ); *ne = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (  u0       ); *e  = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 3 * (  u0 + -u1 ); *se = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (        u1 ); *n  = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 0                ; *c  = EQUILIBRIUM(rho, 1./3 , cu, usqr ); }
    { double cu = 3 * (       -u1 ); *s  = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 3 * ( -u0 +  u1 ); *nw = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * ( -u0       ); *w  = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 3 * ( -u0 + -u1 ); *sw = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (  u0 +  u2 ); *te = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (  u1 +  u2 ); *tn = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (        u2 ); *tc = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 3 * ( -u1 +  u2 ); *ts = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * ( -u0 +  u2 ); *tw = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (  u0 + -u2 ); *be = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (  u1 + -u2 ); *bn = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * (       -u2 ); *bc = EQUILIBRIUM(rho, 1./18, cu, usqr ); }
    { double cu = 3 * ( -u1 + -u2 ); *bs = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
    { double cu = 3 * ( -u0 + -u2 ); *bw = EQUILIBRIUM(rho, 1./36, cu, usqr ); }
}

__device__ static void macroscopic(double ne, double e, double se, double n, double c, double s, double nw, double w, double sw,
                                   double te, double tn, double tc, double ts, double tw, 
                                   double be, double bn, double bc, double bs, double bw, 
                                   double* rho, double* u0, double* u1, double* u2)
{   
    *rho = ne + e  + se + n  + c  + s  + nw + w  + sw + te + tn + tc + ts + tw + be + bn + bc + bs + bw;
    *u0 = (ne + e  + se - nw - w  - sw + te - tw + be - bw) / *rho;
    *u1 = (ne - se + n  - s  + nw - sw + tn - ts + bn - bs) / *rho;
    *u2 = (te + tn + tc + ts + tw - be - bn - bc - bs - bw) / *rho;
}

__global__ void lbm_computation(lbm_vars d_vars, lbm_lattices f0, lbm_lattices f1, size_t nx, size_t ny, size_t nz, double omega)
{
    int tix = threadIdx.x;
    for (int z = blockIdx.z; z < nz; z+=gridDim.z) {
        for (int y = blockIdx.y; y < ny; y+=gridDim.y) {
            for (int x = threadIdx.x + blockIdx.x * blockDim.x; x < nx; x += blockDim.x * gridDim.x) {
                size_t gi = IDX(x,y,z,nx,ny,nz);

                double fin_ne, fin_e, fin_se, fin_n, fin_c, fin_s, fin_nw, fin_w, fin_sw,
                       fin_te, fin_tn, fin_tc, fin_ts, fin_tw, 
                       fin_be, fin_bn, fin_bc, fin_bs, fin_bw;
                double fout_ne, fout_e, fout_se, fout_n, fout_c, fout_s, fout_nw, fout_w, fout_sw,
                       fout_te, fout_tn, fout_tc, fout_ts, fout_tw, 
                       fout_be, fout_bn, fout_bc, fout_bs, fout_bw;

                fin_ne = f0.ne[gi];
                fin_e  = f0.e [gi];
                fin_se = f0.se[gi];
                fin_n  = f0.n [gi];
                fin_c  = f0.c [gi];
                fin_s  = f0.s [gi];
                fin_nw = f0.nw[gi];
                fin_w  = f0.w [gi];
                fin_sw = f0.sw[gi];
                fin_te = f0.te[gi];
                fin_tn = f0.tn[gi];
                fin_tc = f0.tc[gi];
                fin_ts = f0.ts[gi];
                fin_tw = f0.tw[gi];
                fin_be = f0.be[gi];
                fin_bn = f0.bn[gi];
                fin_bc = f0.bc[gi];
                fin_bs = f0.bs[gi];
                fin_bw = f0.bw[gi];

                // Compute macroscopic variables, density and velocity
                double rho, u0, u1, u2;
                macroscopic(fin_ne, fin_e, fin_se, fin_n, fin_c, fin_s, fin_nw, fin_w, fin_sw, 
                            fin_te, fin_tn, fin_tc, fin_ts, fin_tw, 
                            fin_be, fin_bn, fin_bc, fin_bs, fin_bw,
                            &rho, &u0, &u1, &u2);

                // Compute equilibrium
                double feq_ne, feq_e, feq_se, feq_n, feq_c, feq_s, feq_nw, feq_w, feq_sw, 
                       feq_te, feq_tn, feq_tc, feq_ts, feq_tw, 
                       feq_be, feq_bn, feq_bc, feq_bs, feq_bw;
                equilibrium(&feq_ne, &feq_e, &feq_se, &feq_n, &feq_c, &feq_s, &feq_nw, &feq_w, &feq_sw, 
                            &feq_te, &feq_tn, &feq_tc, &feq_ts, &feq_tw, 
                            &feq_be, &feq_bn, &feq_bc, &feq_bs, &feq_bw, 
                            rho, u0, u1, u2);       

                if (d_vars.obstacles[IDX(x,y,z,nx,ny,nz)]) {
                    // Bounce-back condition for obstacle
                    fout_ne = fin_sw; 
                    fout_e  = fin_w ; 
                    fout_se = fin_nw; 
                    fout_n  = fin_s ; 
                    fout_c  = fin_c ; 
                    fout_s  = fin_n ; 
                    fout_nw = fin_se; 
                    fout_w  = fin_e ; 
                    fout_sw = fin_ne; 

                    fout_te = fin_bw;
                    fout_tn = fin_bs;
                    fout_tc = fin_bc;
                    fout_ts = fin_bn;
                    fout_tw = fin_be;
                    fout_be = fin_tw;
                    fout_bn = fin_ts;
                    fout_bc = fin_tc;
                    fout_bs = fin_tn;
                    fout_bw = fin_tw;

                } else {
                    // Collision step
                    fout_ne = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_ne, - feq_ne)), fin_ne);
                    fout_e  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_e , - feq_e )), fin_e );
                    fout_se = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_se, - feq_se)), fin_se);
                    fout_n  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_n , - feq_n )), fin_n );
                    fout_c  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_c , - feq_c )), fin_c );
                    fout_s  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_s , - feq_s )), fin_s );
                    fout_nw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_nw, - feq_nw)), fin_nw);
                    fout_w  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_w , - feq_w )), fin_w );
                    fout_sw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_sw, - feq_sw)), fin_sw);

                    fout_te = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_te, - feq_te)), fin_te);
                    fout_tn = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tn, - feq_tn)), fin_tn);
                    fout_tc = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tc, - feq_tc)), fin_tc);
                    fout_ts = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_ts, - feq_ts)), fin_ts);
                    fout_tw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tw, - feq_tw)), fin_tw);
                    fout_be = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_be, - feq_be)), fin_be);
                    fout_bn = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bn, - feq_bn)), fin_bn);
                    fout_bc = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bc, - feq_bc)), fin_bc);
                    fout_bs = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bs, - feq_bs)), fin_bs);
                    fout_bw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bw, - feq_bw)), fin_bw);
                }

                // STREAMING

                // shared variables for in-block propagation
                __shared__ double fo_E [BLOCK_SIZE];
                __shared__ double fo_W [BLOCK_SIZE];
                __shared__ double fo_SE[BLOCK_SIZE];
                __shared__ double fo_SW[BLOCK_SIZE];
                __shared__ double fo_NE[BLOCK_SIZE];
                __shared__ double fo_NW[BLOCK_SIZE];

                // Center 'propagation' (global memory)
                f1.c[gi] = fout_c;

                // N + S propagation (global memory)
                f1.s[IDX(x, y-1, z, nx,ny,nz)] = fout_s;
                f1.n[IDX(x, y+1, z, nx,ny,nz)] = fout_n;

                // E propagation in shared memory
                if (tix < blockDim.x-1 && x < nx-1) {
                    fo_E [tix+1] = fout_e;
                    fo_NE[tix+1] = fout_ne;
                    fo_SE[tix+1] = fout_se;
                // E propagation in global memory (at block boundary)
                } else {
                    f1.e [IDX(x+1, y  , z, nx,ny,nz)] = fout_e;
                    f1.se[IDX(x+1, y-1, z, nx,ny,nz)] = fout_se;
                    f1.ne[IDX(x+1, y+1, z, nx,ny,nz)] = fout_ne;
                }

                // W propagation in shared memory
                if (tix > 0) {
                    fo_W [tix-1] = fout_w;
                    fo_NW[tix-1] = fout_nw;
                    fo_SW[tix-1] = fout_sw;
                // W propagation in global memory (at block boundary)
                } else {
                    f1.w [IDX(x-1, y  , z, nx,ny,nz)] = fout_w;
                    f1.sw[IDX(x-1, y-1, z, nx,ny,nz)] = fout_sw;
                    f1.nw[IDX(x-1, y+1, z, nx,ny,nz)] = fout_nw;
                }

                // Top and Bottom propagation (global memory)
                f1.te[IDX(x+1, y  , z+1, nx,ny,nz)] = fout_te;
                f1.tn[IDX(x  , y+1, z+1, nx,ny,nz)] = fout_tn;
                f1.tc[IDX(x  , y  , z+1, nx,ny,nz)] = fout_tc;
                f1.ts[IDX(x  , y-1, z+1, nx,ny,nz)] = fout_ts;
                f1.tw[IDX(x-1, y  , z+1, nx,ny,nz)] = fout_tw;
                f1.be[IDX(x+1, y  , z-1, nx,ny,nz)] = fout_be;
                f1.bn[IDX(x  , y+1, z-1, nx,ny,nz)] = fout_bn;
                f1.bc[IDX(x  , y  , z-1, nx,ny,nz)] = fout_bc;
                f1.bs[IDX(x  , y-1, z-1, nx,ny,nz)] = fout_bs;
                f1.bw[IDX(x-1, y  , z-1, nx,ny,nz)] = fout_bw;

                __syncthreads();

                // the leftmost thread is not updated in this block
                if (tix > 0) {
                    f1.e [gi                      ] = fo_E [tix];
                    f1.se[IDX(x, y-1, z, nx,ny,nz)] = fo_SE[tix];
                    f1.ne[IDX(x, y+1, z, nx,ny,nz)] = fo_NE[tix];
                }

                // the rightmost thread is not updated in this block
                if (tix < blockDim.x-1 && x < nx-1) {
                    f1.w [gi                      ] = fo_W [tix];
                    f1.sw[IDX(x, y-1, z, nx,ny,nz)] = fo_SW[tix];
                    f1.nw[IDX(x, y+1, z, nx,ny,nz)] = fo_NW[tix];
                }

                __syncthreads(); // only nessessary when NX % BLOCK_SIZE != 0 
           }
        }
    }
}

void lbm_lattices_alloc(lbm_lattices* lat, size_t nl) 
{
    lat->ne = (double*) malloc ( sizeof(double)*nl );
    lat->e  = (double*) malloc ( sizeof(double)*nl );
    lat->se = (double*) malloc ( sizeof(double)*nl );
    lat->n  = (double*) malloc ( sizeof(double)*nl );
    lat->c  = (double*) malloc ( sizeof(double)*nl );
    lat->s  = (double*) malloc ( sizeof(double)*nl );
    lat->nw = (double*) malloc ( sizeof(double)*nl );
    lat->w  = (double*) malloc ( sizeof(double)*nl );
    lat->sw = (double*) malloc ( sizeof(double)*nl );
    lat->te = (double*) malloc ( sizeof(double)*nl );
    lat->tn = (double*) malloc ( sizeof(double)*nl );
    lat->tc = (double*) malloc ( sizeof(double)*nl );
    lat->ts = (double*) malloc ( sizeof(double)*nl );
    lat->tw = (double*) malloc ( sizeof(double)*nl );
    lat->be = (double*) malloc ( sizeof(double)*nl );
    lat->bn = (double*) malloc ( sizeof(double)*nl );
    lat->bc = (double*) malloc ( sizeof(double)*nl );
    lat->bs = (double*) malloc ( sizeof(double)*nl );
    lat->bw = (double*) malloc ( sizeof(double)*nl );
}

void lbm_lattices_cuda_alloc(lbm_lattices* lat, size_t nl) 
{
    HANDLE_ERROR(hipMalloc(&lat->ne, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->e , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->se, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->n , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->c , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->s , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->nw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->w , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->sw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->te, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tn, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tc, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->ts, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->be, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bn, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bc, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bs, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bw, sizeof(double)*nl ));
}

void lbm_lattices_dealloc(lbm_lattices* lat) 
{
    free(lat->ne);
    free(lat->e );
    free(lat->se);
    free(lat->n );
    free(lat->c );
    free(lat->s );
    free(lat->nw);
    free(lat->w );
    free(lat->sw);
    free(lat->te);
    free(lat->tn);
    free(lat->tc);
    free(lat->ts);
    free(lat->tw);
    free(lat->be);
    free(lat->bn);
    free(lat->bc);
    free(lat->bs);
    free(lat->bw);
}

void lbm_lattices_cuda_dealloc(lbm_lattices* lat) 
{
    HANDLE_ERROR(hipFree(lat->ne));
    HANDLE_ERROR(hipFree(lat->e ));
    HANDLE_ERROR(hipFree(lat->se));
    HANDLE_ERROR(hipFree(lat->n ));
    HANDLE_ERROR(hipFree(lat->c ));
    HANDLE_ERROR(hipFree(lat->s ));
    HANDLE_ERROR(hipFree(lat->nw));
    HANDLE_ERROR(hipFree(lat->w ));
    HANDLE_ERROR(hipFree(lat->sw));
    HANDLE_ERROR(hipFree(lat->te));
    HANDLE_ERROR(hipFree(lat->tn));
    HANDLE_ERROR(hipFree(lat->tc));
    HANDLE_ERROR(hipFree(lat->ts));
    HANDLE_ERROR(hipFree(lat->tw));
    HANDLE_ERROR(hipFree(lat->be));
    HANDLE_ERROR(hipFree(lat->bn));
    HANDLE_ERROR(hipFree(lat->bc));
    HANDLE_ERROR(hipFree(lat->bs));
    HANDLE_ERROR(hipFree(lat->bw));
}

void lbm_vars_alloc(lbm_vars* vars, size_t nl)
{
    vars->obstacles = (bool*) malloc( sizeof(bool)*nl);
    lbm_lattices_alloc(&vars->f0, nl);
    lbm_lattices_alloc(&vars->f1, nl);
}

void lbm_vars_cuda_alloc(lbm_vars* vars, size_t nl)
{
    HANDLE_ERROR(hipMalloc(&vars->obstacles, sizeof(bool)*nl));
    lbm_lattices_cuda_alloc(&vars->f0, nl);
    lbm_lattices_cuda_alloc(&vars->f1, nl);
}

void lbm_vars_dealloc(lbm_vars* vars)
{
    free(vars->obstacles);
    lbm_lattices_dealloc(&vars->f0);
    lbm_lattices_dealloc(&vars->f1);
}

void lbm_vars_cuda_dealloc(lbm_vars* vars)
{
    HANDLE_ERROR(hipFree(vars->obstacles));
    lbm_lattices_cuda_dealloc(&vars->f0);
    lbm_lattices_cuda_dealloc(&vars->f1);
}

lbm_simulation* lbm_simulation_create(size_t nx, size_t ny, size_t nz, double omega)
{
    lbm_simulation* lbm_sim = (lbm_simulation*) malloc (sizeof(lbm_simulation));
    size_t nl = nx*ny*nz;
    lbm_sim->nx = nx;
    lbm_sim->ny = ny;
    lbm_sim->nz = nz;
    lbm_sim->omega = omega;
           
    // Initialization of the populations at equilibrium with the given velocity.
    lbm_sim->switch_f0_f1 = false;

    lbm_vars_cuda_alloc(&lbm_sim->d_vars, nl);

//    HANDLE_ERROR(hipMemcpy(lbm_sim->d_vars.obstacles, lbm_sim->h_vars.obstacles, sizeof(bool)*nl, hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    dim3 dimComputationGrid(max((unsigned long)1, (unsigned long)nx/BLOCK_SIZE), min((unsigned long)ny, (unsigned long)prop.maxGridSize[1]), min((unsigned long)nz, (unsigned long)prop.maxGridSize[2]));
    dim3 dimComputationBlock(BLOCK_SIZE);
    lbm_sim->dimComputationGrid = dimComputationGrid;
    lbm_sim->dimComputationBlock = dimComputationBlock;

    dim3 dimRightWallGrid(1, min((unsigned long)ny, (unsigned long)prop.maxGridSize[1]));
    dim3 dimRightWallBlock(1);
    lbm_sim->dimRightWallGrid = dimRightWallGrid;
    lbm_sim->dimRightWallBlock = dimRightWallBlock;

    lbm_sim->shared_mem_size = 6 * sizeof(double) * BLOCK_SIZE;

    return lbm_sim;
}

void lbm_simulation_destroy(lbm_simulation* lbm_sim)
{
    lbm_vars_cuda_dealloc(&lbm_sim->d_vars);
    free(lbm_sim);
}

void lbm_simulation_update(lbm_simulation* lbm_sim)
{
    if (lbm_sim->switch_f0_f1) {
        HANDLE_KERNEL_ERROR(lbm_computation<<<lbm_sim->dimComputationGrid, lbm_sim->dimComputationBlock, lbm_sim->shared_mem_size>>>(lbm_sim->d_vars, lbm_sim->d_vars.f1, lbm_sim->d_vars.f0, lbm_sim->nx, lbm_sim->ny, lbm_sim->nz, lbm_sim->omega));
    } else {
        HANDLE_KERNEL_ERROR(lbm_computation<<<lbm_sim->dimComputationGrid, lbm_sim->dimComputationBlock, lbm_sim->shared_mem_size>>>(lbm_sim->d_vars, lbm_sim->d_vars.f0, lbm_sim->d_vars.f1, lbm_sim->nx, lbm_sim->ny, lbm_sim->nz, lbm_sim->omega));
    }

    lbm_sim->switch_f0_f1 = ! lbm_sim->switch_f0_f1;
}

void lbm_lattices_write(lbm_simulation* lbm_sim, lbm_lattices* h_lat, size_t nl)
{
    lbm_lattices* d_lat = lbm_sim->switch_f0_f1 ? &lbm_sim->d_vars.f1 : &lbm_sim->d_vars.f0;

    HANDLE_ERROR(hipMemcpy(d_lat->ne, h_lat->ne, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->e , h_lat->e , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->se, h_lat->se, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->n , h_lat->n , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->c , h_lat->c , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->s , h_lat->s , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->nw, h_lat->nw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->w , h_lat->w , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->sw, h_lat->sw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->te, h_lat->te, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tn, h_lat->tn, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tc, h_lat->tc, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->ts, h_lat->ts, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tw, h_lat->tw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->be, h_lat->be, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bn, h_lat->bn, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bc, h_lat->bc, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bs, h_lat->bs, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bw, h_lat->bw, sizeof(double)*nl, hipMemcpyHostToDevice));
}

void lbm_lattices_read(lbm_simulation* lbm_sim, lbm_lattices* h_lat)
{
    size_t nl = lbm_sim->nx * lbm_sim->ny * lbm_sim->nz;

    lbm_lattices* d_lat = lbm_sim->switch_f0_f1 ? &lbm_sim->d_vars.f1 : &lbm_sim->d_vars.f0;

    HANDLE_ERROR(hipMemcpy(h_lat->ne, d_lat->ne, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->e , d_lat->e , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->se, d_lat->se, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->n , d_lat->n , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->c , d_lat->c , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->s , d_lat->s , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->nw, d_lat->nw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->w , d_lat->w , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->sw, d_lat->sw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->te, d_lat->te, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tn, d_lat->tn, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tc, d_lat->tc, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->ts, d_lat->ts, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tw, d_lat->tw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->be, d_lat->be, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bn, d_lat->bn, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bc, d_lat->bc, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bs, d_lat->bs, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bw, d_lat->bw, sizeof(double)*nl, hipMemcpyDeviceToHost));
}

lbm_lattices* lbm_lattices_create(size_t nl)
{
    lbm_lattices* lat = (lbm_lattices*) malloc(sizeof(lbm_lattices));
    lbm_lattices_alloc(lat, nl);
    return lat;
}

void lbm_lattices_destroy(lbm_lattices* lat)
{
    lbm_lattices_dealloc(lat);
    free(lat);
}

