#include "hip/hip_runtime.h"
/*!
 * \file    lbmFlowAroundCylinder.cu
 * \brief   Cuda version based on lbm_sailfish_hist and lbm_opt1.
 * \author  Adrien Python
 * \date    28.01.2017
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include "lbmcuda.h"

#define BLOCK_SIZE 64

#define SQUARE(a) ((a)*(a))
#define GPU_SQUARE(a) __dmul_rn(a,a)

typedef struct {
    lbm_u u;
    lbm_lattices f0;
    lbm_lattices f1;
} lbm_vars;

#define HANDLE_ERROR(ans) (handleError((ans), __FILE__, __LINE__))
inline void handleError(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(EXIT_FAILURE);
   }
}

#define HANDLE_KERNEL_ERROR(...) \
do {                                         \
    __VA_ARGS__;                             \
/*    HANDLE_ERROR( hipPeekAtLastError() );  */ \
/*    HANDLE_ERROR( hipDeviceSynchronize() );*/ \
} while(0)


#define EQUILIBRIUM(rho, t, cu, usqr) __dmul_rn(__dmul_rn(rho, (t)), __dadd_rn(__dadd_rn(__dadd_rn(1, cu) , __dmul_rn(0.5, GPU_SQUARE(cu))), - usqr) )


__device__ static void equilibrium(double* ne, double* e, double* se, double* n, double* c, double* s, double* nw, double* w, double* sw, 
                                     double* te, double* tn, double* tc, double* ts, double* tw,
                                     double* be, double* bn, double* bc, double* bs, double* bw,
                                     double rho, double u0, double u1, double u2)
{
    double usqr = __dmul_rn(3./2, __dadd_rn( __dadd_rn( GPU_SQUARE(u0), GPU_SQUARE(u1)), GPU_SQUARE(u2) ));

    { double cu = 3 * (  u0 +  u1 ); *ne = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (  u0       ); *e  = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 3 * (  u0 + -u1 ); *se = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (        u1 ); *n  = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 0                ; *c  = EQUILIBRIUM(rho, 1./3 , cu, usqr ); } 
    { double cu = 3 * (       -u1 ); *s  = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 3 * ( -u0 +  u1 ); *nw = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * ( -u0       ); *w  = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 3 * ( -u0 + -u1 ); *sw = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (  u0 +  u2 ); *te = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (  u1 +  u2 ); *tn = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (        u2 ); *tc = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 3 * ( -u1 +  u2 ); *ts = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * ( -u0 +  u2 ); *tw = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (  u0 + -u2 ); *be = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (  u1 + -u2 ); *bn = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * (       -u2 ); *bc = EQUILIBRIUM(rho, 1./18, cu, usqr ); } 
    { double cu = 3 * ( -u1 + -u2 ); *bs = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
    { double cu = 3 * ( -u0 + -u2 ); *bw = EQUILIBRIUM(rho, 1./36, cu, usqr ); } 
}

__device__ static void macroscopic(double ne, double e, double se, double n, double c, double s, double nw, double w, double sw,
                                   double te, double tn, double tc, double ts, double tw, 
                                   double be, double bn, double bc, double bs, double bw, 
                                   double* rho, double* u0, double* u1, double* u2)
{   
    *rho = ne + e  + se + n  + c  + s  + nw + w  + sw + te + tn + tc + ts + tw + be + bn + bc + bs + bw;
    *u0 = (ne + e  + se - nw - w  - sw + te - tw + be - bw) / *rho;
    *u1 = (ne - se + n  - s  + nw - sw + tn - ts + bn - bs) / *rho;
    *u2 = (te + tn + tc + ts + tw - be - bn - bc - bs - bw) / *rho;
}

__global__ void lbm_computation(lbm_vars d_vars, lbm_lattices f0, lbm_lattices f1, size_t nx, size_t ny, size_t nz, double omega)
{
    for (int z = blockIdx.z; z < nz; z+=gridDim.z) {
        for (int y = blockIdx.y; y < ny; y+=gridDim.y) {
            for (int x = threadIdx.x + blockIdx.x * blockDim.x; x < nx; x += blockDim.x * gridDim.x) {
                size_t gi = IDX(x,y,z,nx,ny,nz);

                double fin_ne, fin_e, fin_se, fin_n, fin_c, fin_s, fin_nw, fin_w, fin_sw,
                       fin_te, fin_tn, fin_tc, fin_ts, fin_tw, 
                       fin_be, fin_bn, fin_bc, fin_bs, fin_bw;
                double fout_ne, fout_e, fout_se, fout_n, fout_c, fout_s, fout_nw, fout_w, fout_sw,
                       fout_te, fout_tn, fout_tc, fout_ts, fout_tw, 
                       fout_be, fout_bn, fout_bc, fout_bs, fout_bw;
                double feq_ne, feq_e, feq_se, feq_n, feq_c, feq_s, feq_nw, feq_w, feq_sw, 
                       feq_te, feq_tn, feq_tc, feq_ts, feq_tw, 
                       feq_be, feq_bn, feq_bc, feq_bs, feq_bw;

                fin_ne = f0.ne[gi];
                fin_e  = f0.e [gi];
                fin_se = f0.se[gi];
                fin_n  = f0.n [gi];
                fin_c  = f0.c [gi];
                fin_s  = f0.s [gi];
                fin_nw = f0.nw[gi];
                fin_w  = f0.w [gi];
                fin_sw = f0.sw[gi];
                fin_te = f0.te[gi];
                fin_tn = f0.tn[gi];
                fin_tc = f0.tc[gi];
                fin_ts = f0.ts[gi];
                fin_tw = f0.tw[gi];
                fin_be = f0.be[gi];
                fin_bn = f0.bn[gi];
                fin_bc = f0.bc[gi];
                fin_bs = f0.bs[gi];
                fin_bw = f0.bw[gi];

                // Compute macroscopic variables, density and velocity
                double rho, u0, u1, u2;
                macroscopic(fin_ne, fin_e, fin_se, fin_n, fin_c, fin_s, fin_nw, fin_w, fin_sw, 
                            fin_te, fin_tn, fin_tc, fin_ts, fin_tw, 
                            fin_be, fin_bn, fin_bc, fin_bs, fin_bw,
                            &rho, &u0, &u1, &u2);

                // Compute equilibrium
                equilibrium(&feq_ne, &feq_e, &feq_se, &feq_n, &feq_c, &feq_s, &feq_nw, &feq_w, &feq_sw, 
                            &feq_te, &feq_tn, &feq_tc, &feq_ts, &feq_tw, 
                            &feq_be, &feq_bn, &feq_bc, &feq_bs, &feq_bw, 
                            rho, u0, u1, u2);       

                d_vars.u.u0[gi] = u0;
                d_vars.u.u1[gi] = u1;
                d_vars.u.u2[gi] = u2;

                 // Collision step
                fout_ne = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_ne, - feq_ne)), fin_ne);
                fout_e  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_e , - feq_e )), fin_e );
                fout_se = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_se, - feq_se)), fin_se);
                fout_n  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_n , - feq_n )), fin_n );
                fout_c  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_c , - feq_c )), fin_c );
                fout_s  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_s , - feq_s )), fin_s );
                fout_nw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_nw, - feq_nw)), fin_nw);
                fout_w  = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_w , - feq_w )), fin_w );
                fout_sw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_sw, - feq_sw)), fin_sw);

                fout_te = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_te, - feq_te)), fin_te);
                fout_tn = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tn, - feq_tn)), fin_tn);
                fout_tc = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tc, - feq_tc)), fin_tc);
                fout_ts = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_ts, - feq_ts)), fin_ts);
                fout_tw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_tw, - feq_tw)), fin_tw);
                fout_be = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_be, - feq_be)), fin_be);
                fout_bn = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bn, - feq_bn)), fin_bn);
                fout_bc = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bc, - feq_bc)), fin_bc);
                fout_bs = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bs, - feq_bs)), fin_bs);
                fout_bw = __dadd_rn(__dmul_rn(-omega, __dadd_rn(fin_bw, - feq_bw)), fin_bw);

                // Streaming
                f1.c [IDX(x  , y  , z  , nx,ny,nz)] = fout_c;
                f1.s [IDX(x  , y-1, z  , nx,ny,nz)] = fout_s;
                f1.n [IDX(x  , y+1, z  , nx,ny,nz)] = fout_n;
                f1.e [IDX(x+1, y  , z  , nx,ny,nz)] = fout_e;
                f1.se[IDX(x+1, y-1, z  , nx,ny,nz)] = fout_se;
                f1.ne[IDX(x+1, y+1, z  , nx,ny,nz)] = fout_ne;
                f1.w [IDX(x-1, y  , z  , nx,ny,nz)] = fout_w;
                f1.sw[IDX(x-1, y-1, z  , nx,ny,nz)] = fout_sw;
                f1.nw[IDX(x-1, y+1, z  , nx,ny,nz)] = fout_nw;
                f1.te[IDX(x+1, y  , z+1, nx,ny,nz)] = fout_te;
                f1.tn[IDX(x  , y+1, z+1, nx,ny,nz)] = fout_tn;
                f1.tc[IDX(x  , y  , z+1, nx,ny,nz)] = fout_tc;
                f1.ts[IDX(x  , y-1, z+1, nx,ny,nz)] = fout_ts;
                f1.tw[IDX(x-1, y  , z+1, nx,ny,nz)] = fout_tw;
                f1.be[IDX(x+1, y  , z-1, nx,ny,nz)] = fout_be;
                f1.bn[IDX(x  , y+1, z-1, nx,ny,nz)] = fout_bn;
                f1.bc[IDX(x  , y  , z-1, nx,ny,nz)] = fout_bc;
                f1.bs[IDX(x  , y-1, z-1, nx,ny,nz)] = fout_bs;
                f1.bw[IDX(x-1, y  , z-1, nx,ny,nz)] = fout_bw;
            }
        }
    }
}

struct lbm_simulation{
    lbm_vars h_vars, d_vars;
    dim3 dimComputationGrid, dimComputationBlock;
    size_t shared_mem_size;
    bool switch_f0_f1;
    size_t nx, ny, nz, nl;
    double omega;
};

void lbm_lattices_alloc(lbm_lattices* lat, size_t nl) {
    lat->ne = (double*) malloc ( sizeof(double)*nl );
    lat->e  = (double*) malloc ( sizeof(double)*nl );
    lat->se = (double*) malloc ( sizeof(double)*nl );
    lat->n  = (double*) malloc ( sizeof(double)*nl );
    lat->c  = (double*) malloc ( sizeof(double)*nl );
    lat->s  = (double*) malloc ( sizeof(double)*nl );
    lat->nw = (double*) malloc ( sizeof(double)*nl );
    lat->w  = (double*) malloc ( sizeof(double)*nl );
    lat->sw = (double*) malloc ( sizeof(double)*nl );
    lat->te = (double*) malloc ( sizeof(double)*nl );
    lat->tn = (double*) malloc ( sizeof(double)*nl );
    lat->tc = (double*) malloc ( sizeof(double)*nl );
    lat->ts = (double*) malloc ( sizeof(double)*nl );
    lat->tw = (double*) malloc ( sizeof(double)*nl );
    lat->be = (double*) malloc ( sizeof(double)*nl );
    lat->bn = (double*) malloc ( sizeof(double)*nl );
    lat->bc = (double*) malloc ( sizeof(double)*nl );
    lat->bs = (double*) malloc ( sizeof(double)*nl );
    lat->bw = (double*) malloc ( sizeof(double)*nl );
}

void lbm_lattices_cuda_alloc(lbm_lattices* lat, size_t nl) {
    HANDLE_ERROR(hipMalloc(&lat->ne, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->e , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->se, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->n , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->c , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->s , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->nw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->w , sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->sw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->te, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tn, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tc, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->ts, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->tw, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->be, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bn, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bc, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bs, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&lat->bw, sizeof(double)*nl ));
}

void lbm_lattices_dealloc(lbm_lattices* lat) {
    free(lat->ne);
    free(lat->e );
    free(lat->se);
    free(lat->n );
    free(lat->c );
    free(lat->s );
    free(lat->nw);
    free(lat->w );
    free(lat->sw);
    free(lat->te);
    free(lat->tn);
    free(lat->tc);
    free(lat->ts);
    free(lat->tw);
    free(lat->be);
    free(lat->bn);
    free(lat->bc);
    free(lat->bs);
    free(lat->bw);
}

void lbm_lattices_cuda_dealloc(lbm_lattices* lat) {
    HANDLE_ERROR(hipFree(lat->ne));
    HANDLE_ERROR(hipFree(lat->e ));
    HANDLE_ERROR(hipFree(lat->se));
    HANDLE_ERROR(hipFree(lat->n ));
    HANDLE_ERROR(hipFree(lat->c ));
    HANDLE_ERROR(hipFree(lat->s ));
    HANDLE_ERROR(hipFree(lat->nw));
    HANDLE_ERROR(hipFree(lat->w ));
    HANDLE_ERROR(hipFree(lat->sw));
    HANDLE_ERROR(hipFree(lat->te));
    HANDLE_ERROR(hipFree(lat->tn));
    HANDLE_ERROR(hipFree(lat->tc));
    HANDLE_ERROR(hipFree(lat->ts));
    HANDLE_ERROR(hipFree(lat->tw));
    HANDLE_ERROR(hipFree(lat->be));
    HANDLE_ERROR(hipFree(lat->bn));
    HANDLE_ERROR(hipFree(lat->bc));
    HANDLE_ERROR(hipFree(lat->bs));
    HANDLE_ERROR(hipFree(lat->bw));
}

void lbm_u_alloc(lbm_u* u, size_t nl) {
    u->u0 = (double*) malloc( sizeof(double)*nl );
    u->u1 = (double*) malloc( sizeof(double)*nl );
    u->u2 = (double*) malloc( sizeof(double)*nl );
}

void lbm_u_cuda_alloc(lbm_u* u, size_t nl) {
    HANDLE_ERROR(hipMalloc(&u->u0, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&u->u1, sizeof(double)*nl ));
    HANDLE_ERROR(hipMalloc(&u->u2, sizeof(double)*nl ));
}

void lbm_u_dealloc(lbm_u* u) {
    free(u->u0);
    free(u->u1);
    free(u->u2);
}

void lbm_u_cuda_dealloc(lbm_u* u) {
    HANDLE_ERROR(hipFree(u->u0));
    HANDLE_ERROR(hipFree(u->u1));
    HANDLE_ERROR(hipFree(u->u2));
}

void lbm_vars_alloc(lbm_vars* vars, size_t nl)
{
    lbm_u_alloc(&vars->u, nl);
    lbm_lattices_alloc(&vars->f0, nl);
    lbm_lattices_alloc(&vars->f1, nl);
}

void lbm_vars_cuda_alloc(lbm_vars* vars, size_t nl)
{
    lbm_u_cuda_alloc(&vars->u, nl);
    lbm_lattices_cuda_alloc(&vars->f0, nl);
    lbm_lattices_cuda_alloc(&vars->f1, nl);
}

void lbm_vars_dealloc(lbm_vars* vars)
{
    lbm_u_dealloc(&vars->u);
    lbm_lattices_dealloc(&vars->f0);
    lbm_lattices_dealloc(&vars->f1);
}

void lbm_vars_cuda_dealloc(lbm_vars* vars)
{
    lbm_u_cuda_dealloc(&vars->u);
    lbm_lattices_cuda_dealloc(&vars->f0);
    lbm_lattices_cuda_dealloc(&vars->f1);
}

lbm_simulation* lbm_simulation_create(size_t nx, size_t ny, size_t nz, double omega)
{
    lbm_simulation* lbm_sim = (lbm_simulation*) malloc (sizeof(lbm_simulation));

    lbm_sim->nx = nx;
    lbm_sim->ny = ny;
    lbm_sim->nz = nz;
    lbm_sim->nl = nx * ny * nz;

    lbm_sim->omega = omega;
   
    lbm_vars_alloc(&lbm_sim->h_vars, lbm_sim->nl);
    
    lbm_sim->switch_f0_f1 = false;

    lbm_vars_cuda_alloc(&lbm_sim->d_vars, lbm_sim->nl);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    dim3 dimComputationGrid(max((unsigned long)1, (unsigned long)nx/BLOCK_SIZE), min((unsigned long)ny, (unsigned long)prop.maxGridSize[1]), min((unsigned long)nz, (unsigned long)prop.maxGridSize[2]));
    dim3 dimComputationBlock(BLOCK_SIZE);
    lbm_sim->dimComputationGrid = dimComputationGrid;
    lbm_sim->dimComputationBlock = dimComputationBlock;

    lbm_sim->shared_mem_size = 0; //6 * sizeof(double) * BLOCK_SIZE;

    if ( hipDeviceSetCacheConfig (hipFuncCachePreferL1) != hipSuccess)
        fprintf(stderr, "hipFuncSetCacheConfig failed\n");

    return lbm_sim;
}

void lbm_simulation_destroy(lbm_simulation* lbm_sim)
{
    lbm_vars_dealloc(&lbm_sim->h_vars);
    lbm_vars_cuda_dealloc(&lbm_sim->d_vars);
    free(lbm_sim);
}
void lbm_simulation_update(lbm_simulation* lbm_sim)
{
    if (lbm_sim->switch_f0_f1) {
        HANDLE_KERNEL_ERROR(lbm_computation<<<lbm_sim->dimComputationGrid, lbm_sim->dimComputationBlock, lbm_sim->shared_mem_size>>>(lbm_sim->d_vars, lbm_sim->d_vars.f1, lbm_sim->d_vars.f0, lbm_sim->nx, lbm_sim->ny, lbm_sim->nz, lbm_sim->omega));
    } else {
        HANDLE_KERNEL_ERROR(lbm_computation<<<lbm_sim->dimComputationGrid, lbm_sim->dimComputationBlock, lbm_sim->shared_mem_size>>>(lbm_sim->d_vars, lbm_sim->d_vars.f0, lbm_sim->d_vars.f1, lbm_sim->nx, lbm_sim->ny, lbm_sim->nz, lbm_sim->omega));
    }

    lbm_sim->switch_f0_f1 = ! lbm_sim->switch_f0_f1;
}

void lbm_lattices_read(lbm_simulation* lbm_sim, lbm_lattices* h_lat)
{
    size_t nl = lbm_sim->nl;

    lbm_lattices* d_lat = lbm_sim->switch_f0_f1 ? &lbm_sim->d_vars.f1 : &lbm_sim->d_vars.f0;

    HANDLE_ERROR(hipMemcpy(h_lat->ne, d_lat->ne, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->e , d_lat->e , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->se, d_lat->se, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->n , d_lat->n , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->c , d_lat->c , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->s , d_lat->s , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->nw, d_lat->nw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->w , d_lat->w , sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->sw, d_lat->sw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->te, d_lat->te, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tn, d_lat->tn, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tc, d_lat->tc, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->ts, d_lat->ts, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->tw, d_lat->tw, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->be, d_lat->be, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bn, d_lat->bn, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bc, d_lat->bc, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bs, d_lat->bs, sizeof(double)*nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(h_lat->bw, d_lat->bw, sizeof(double)*nl, hipMemcpyDeviceToHost));
}

void lbm_lattices_write(lbm_simulation* lbm_sim, lbm_lattices* h_lat)
{
    size_t nl = lbm_sim->nl;

    lbm_lattices* d_lat = lbm_sim->switch_f0_f1 ? &lbm_sim->d_vars.f1 : &lbm_sim->d_vars.f0;

    HANDLE_ERROR(hipMemcpy(d_lat->ne, h_lat->ne, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->e , h_lat->e , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->se, h_lat->se, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->n , h_lat->n , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->c , h_lat->c , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->s , h_lat->s , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->nw, h_lat->nw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->w , h_lat->w , sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->sw, h_lat->sw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->te, h_lat->te, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tn, h_lat->tn, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tc, h_lat->tc, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->ts, h_lat->ts, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->tw, h_lat->tw, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->be, h_lat->be, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bn, h_lat->bn, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bc, h_lat->bc, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bs, h_lat->bs, sizeof(double)*nl, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_lat->bw, h_lat->bw, sizeof(double)*nl, hipMemcpyHostToDevice));
}


void lbm_u_read(lbm_simulation* lbm_sim, lbm_u* u)
{
    HANDLE_ERROR(hipMemcpy(u->u0, lbm_sim->d_vars.u.u0, sizeof(double)*lbm_sim->nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(u->u1, lbm_sim->d_vars.u.u1, sizeof(double)*lbm_sim->nl, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(u->u2, lbm_sim->d_vars.u.u2, sizeof(double)*lbm_sim->nl, hipMemcpyDeviceToHost));
}

lbm_lattices* lbm_lattices_create(size_t nl)
{
    lbm_lattices* lat = (lbm_lattices*) malloc(sizeof(lbm_lattices));
    lbm_lattices_alloc(lat, nl);
    return lat;
}

void lbm_lattices_destroy(lbm_lattices* lat)
{
    lbm_lattices_dealloc(lat);
    free(lat);
}

lbm_u* lbm_u_create(size_t nx, size_t ny, size_t nz)
{
    lbm_u* u = (lbm_u*) malloc(sizeof(lbm_u));
    lbm_u_alloc(u, nx*ny*nz);
    return u;
}

void lbm_u_destroy(lbm_u* u)
{
    lbm_u_dealloc(u);
    free(u);
}
