#include "hip/hip_runtime.h"
/*!
 * \file    lbmFlowAroundCylinder.cu
 * \brief   GPU (Cuda) and CPU version running the same code for floating point computation debugging...
 * \author  Adrien Python
 * \date    22.01.2017
 */

#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <libgen.h>
#include <pgm.h>

#define RE       220.0       // Reynolds number
#define NX       100//420         // Numer of lattice nodes (width)
#define NY       10 //180         // Numer of lattice nodes (height)
#define LY       ((NY) - 1)  // Height of the domain in lattice units
#define CX       ((NX) / 4)  // X coordinates of the cylinder
#define CY       ((NY) / 2)  // Y coordinates of the cylinder
#define R        ((NY) / 9)  // Cylinder radius
#define ULB      0.04        // Velocity in lattice units
#define NULB     ((ULB) * (R) / (RE))   // Viscoscity in lattice units
#define OMEGA    ((double)1. / (3*(NULB)+0.5))  // Relaxation parameter

#define SQUARE(a) ((a)*(a))
#define GPU_SQUARE(a) (__dmul_rn(a,a))

typedef enum { OUT_FIN, OUT_IMG, OUT_UNK } out_mode;

typedef struct {
    bool obstacles[NX][NY];  // Should reside in lbm_consts but is too big for constant memory
    double u[NX][NY][2];
    double feq[NX][NY][9];
    double fin[NX][NY][9];
    double fout[NX][NY][9];
    double rho[NX][NY];
    double vel[NX][NY][2];
} lbm_vars;

typedef struct {
    size_t col[3][3];
    size_t opp[9];
    ssize_t v[2][9];
    double t[9];
} lbm_consts;

#ifdef COMPUTE_ON_CPU
// Tweak the code to run on CPU
#define hipMalloc(dst_ptr, size)        do { *(dst_ptr) = (lbm_vars*)malloc(size); } while(0)
#define hipMemcpy(dst, src, size, mode) memcpy(dst, src, size)
#define hipMemcpyToSymbol(HIP_SYMBOL(dst), src, size) memcpy(&dst, src, size)
#define hipFree(ptr) free(ptr)

#define HANDLE_ERROR(ans) ans
#define HANDLE_KERNEL_ERROR(...) do { __VA_ARGS__; } while(0)

#define fory(...) for (int y = 0; y < NY; ++y) { __VA_ARGS__; }
#define forxy(...) fory(for (int x = 0; x < NX; ++x) { __VA_ARGS__; })

#define RUN_KERNEL_1D(kernel, th1, ...) fory(kernel(__VA_ARGS__, y))
#define RUN_KERNEL_2D(kernel, th1, th2, ...) forxy(kernel(__VA_ARGS__, x, y))

#else
// Code for GPU usage only
#define HANDLE_ERROR(ans) (handleError((ans), __FILE__, __LINE__))
inline void handleError(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(EXIT_FAILURE);
   }
}

#define HANDLE_KERNEL_ERROR(...) \
do {                                         \
    __VA_ARGS__;                             \
    HANDLE_ERROR( hipPeekAtLastError() );   \
    HANDLE_ERROR( hipDeviceSynchronize() ); \
} while(0)

#define RUN_KERNEL_1D(kernel, th1, ...) HANDLE_KERNEL_ERROR( kernel<<<1, th1>>>(__VA_ARGS__) )
#define RUN_KERNEL_2D(kernel, th1, th2, ...) HANDLE_KERNEL_ERROR( kernel<<<1, th1*th2>>>(__VA_ARGS__) )
#endif

// Constants

#ifndef COMPUTE_ON_CPU
__constant__ 
#endif
lbm_consts d_consts;

const ssize_t V[][9] = {
    { 1, 1, 1, 0, 0, 0,-1,-1,-1 },
    { 1, 0,-1, 1, 0,-1, 1, 0,-1 }
};
const double T[] = { 1./36, 1./9, 1./36, 1./9, 4./9, 1./9, 1./36, 1./9, 1./36 };


/**
 * Setup: cylindrical obstacle and velocity inlet with perturbation
 * Creation of a mask with boolean values, defining the shape of the obstacle.
 */
static void initObstacles(lbm_vars* vars)
{
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            vars->obstacles[x][y] = SQUARE(x-CX) + SQUARE(y-CY) < SQUARE(R);
        }
    }
}

/**
 * Initial velocity profile: almost zero, with a slight perturbation to trigger
 * the instability.
 */
static void initVelocity(lbm_vars* vars)
{
    for (int d = 0; d < 2; d++) {
        for (int x = 0; x < NX; x++) {
            for (int y = 0; y < NY; y++) {
                vars->vel[x][y][d] = (1-d) * ULB * (1 + 0.0001 * sin( y / (double)LY * 2 * M_PI) );
            }
        }
    }
}

static void initRho(lbm_vars* vars)
{
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            vars->rho[x][y] = 1.0;
        }
    }
}

static void initCol(size_t* col, ssize_t v0)
{
    for (int f = 0, i = 0; f < 9 && i < 3; f++) {
        if (V[0][f] == v0) {
            col[i++] = f;
        }
    }
}

static void initOpp(size_t* opp)
{
    for (int f = 0; f < 9; f++) {
        for (int g = 0; g < 9; g++) {
            if (V[0][f] == -V[0][g] && V[1][f] == -V[1][g]) {
                opp[f] = g;
                break;
            }
        }
    }
}

#define EQUILIBRIUM_BODY(v, t) \
    do {                                                                 \
        double usqr = 3./2 * ( SQUARE(u[0]) + SQUARE(u[1]) );            \
                                                                         \
        for (int f = 0; f < 9; f++) {                                    \
            double cu = 3 * ( v[0][f] * u[0] + v[1][f] * u[1] );         \
            feq[f] = rho * t[f] * ( 1 + cu + 0.5 * SQUARE(cu) - usqr );  \
        }                                                                \
    } while(0)

#define GPU_EQUILIBRIUM_BODY(v, t) \
    do {                                                                                \
        double usqr = __dmul_rn(3./2, __dadd_rn( GPU_SQUARE(u[0]), GPU_SQUARE(u[1]) )); \
                                                                                        \
        for (int f = 0; f < 9; f++) {                                                   \
            double cu = 3 * ( v[0][f] * u[0] + v[1][f] * u[1] );                        \
            feq[f] = rho * t[f] * ( 1 + cu + 0.5 * SQUARE(cu) - usqr );                 \
        }                                                                               \
    } while(0)


#ifndef COMPUTE_ON_CPU
__host__ 
#endif
static void h_equilibrium(double* feq, double rho, double* u)
{
    EQUILIBRIUM_BODY(V, T);
}

#ifndef COMPUTE_ON_CPU
__device__ 
#endif
static void d_equilibrium(double* feq, double rho, double* u)
{
#ifdef COMPUTE_ON_CPU
    EQUILIBRIUM_BODY(d_consts.v, d_consts.t);
#else
    GPU_EQUILIBRIUM_BODY(d_consts.v, d_consts.t);
#endif
}

#ifndef COMPUTE_ON_CPU
__device__ 
#endif
static void macroscopic(double* fin, double* rho, double* u)
{
    
    *rho = u[0] = u[1] = 0;

    for (int f = 0; f < 9; f++) {
        *rho += fin[f];

        u[0] += d_consts.v[0][f] * fin[f];
        u[1] += d_consts.v[1][f] * fin[f];
    }
    
    u[0] /= *rho;
    u[1] /= *rho;
}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_right_wall(lbm_vars *d_vars)
#else
void lbm_right_wall(lbm_vars *d_vars, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x;
#endif

    // Right wall: outflow condition.
    for (int i = 0; i < 3; i++) {
        int f = d_consts.col[2][i];
        d_vars->fin[NX-1][y][f] = d_vars->fin[NX-2][y][f]; // TODO Test: retirer condition bord
    }
}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_macro_and_left_wall(lbm_vars *d_vars)
#else
void lbm_macro_and_left_wall(lbm_vars *d_vars, int x, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x / NX;
    int x = threadIdx.x % NX;
#endif

    // Compute macroscopic variables, density and velocity
    macroscopic(d_vars->fin[x][y], &d_vars->rho[x][y], d_vars->u[x][y]);
    
    // Left wall: inflow condition
    for (size_t d = 0; d < 2; d++) {
        d_vars->u[0][y][d] = d_vars->vel[0][y][d]; // TODO: collision!
    }
 }

#ifndef COMPUTE_ON_CPU
__global__ void lbm_density(lbm_vars *d_vars)
#else
void lbm_density(lbm_vars *d_vars, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x;
#endif
    // Calculate the density
    double s2 = 0, s3 = 0;
    for (size_t i = 0; i < 3; i++) {
        s2 += d_vars->fin[0][y][d_consts.col[1][i]];
        s3 += d_vars->fin[0][y][d_consts.col[2][i]];
    }
    d_vars->rho[0][y] = 1./(1 - d_vars->u[0][y][0]) * (s2 + 2*s3);
}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_equilibrium_1(lbm_vars *d_vars)
#else
void lbm_equilibrium_1(lbm_vars *d_vars, int x, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x / NX;
    int x = threadIdx.x % NX;
#endif
   
    // Compute equilibrium
    d_equilibrium(d_vars->feq[x][y], d_vars->rho[x][y], d_vars->u[x][y]);
}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_equilibrium_2(lbm_vars *d_vars)
#else 
void lbm_equilibrium_2(lbm_vars *d_vars, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x;
#endif

    for (size_t i = 0, f = d_consts.col[0][i]; i < 3; f = d_consts.col[0][++i]) {
        d_vars->fin[0][y][f] = d_vars->feq[0][y][f] + d_vars->fin[0][y][d_consts.opp[f]] - d_vars->feq[0][y][d_consts.opp[f]];
    }

}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_collision(lbm_vars *d_vars)
#else
void lbm_collision(lbm_vars *d_vars, int x, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x / NX;
    int x = threadIdx.x % NX;
#endif
    
    for (size_t f = 0; f < 9; f++) {
        if (d_vars->obstacles[x][y]) {
            // Bounce-back condition for obstacle
            d_vars->fout[x][y][f] = d_vars->fin[x][y][d_consts.opp[f]];
        } else {
            // Collision step
#ifdef COMPUTE_ON_CPU
            d_vars->fout[x][y][f] = d_vars->fin[x][y][f] - OMEGA * (d_vars->fin[x][y][f] - d_vars->feq[x][y][f]);
#else
//            d_vars->fout[x][y][f] = __fma_rn(-OMEGA, __dadd_rn(d_vars->fin[x][y][f], - d_vars->feq[x][y][f]), d_vars->fin[x][y][f]); // no change
            d_vars->fout[x][y][f] = __dadd_rn(__dmul_rn(-OMEGA, __dadd_rn(d_vars->fin[x][y][f], - d_vars->feq[x][y][f])), d_vars->fin[x][y][f]);
#endif

        }
    }

}

#ifndef COMPUTE_ON_CPU
__global__ void lbm_streaming(lbm_vars *d_vars)
#else
void lbm_streaming(lbm_vars *d_vars, int x, int y)
#endif
{
#ifndef COMPUTE_ON_CPU
    int y = threadIdx.x / NX;
    int x = threadIdx.x % NX;
#endif

    // Streaming step
    for (size_t f = 0; f < 9; f++) {
        size_t x_dst = (x + NX + d_consts.v[0][f]) % NX;
        size_t y_dst = (y + NY + d_consts.v[1][f]) % NY;
        d_vars->fin[x_dst][y_dst][f] = d_vars->fout[x][y][f];
    }

}

void print_variables(lbm_vars *d_vars, lbm_vars *h_vars, double var[NX][NY][9]) {

    HANDLE_ERROR(hipMemcpy(h_vars, d_vars, sizeof(lbm_vars), hipMemcpyDeviceToHost));

    for (size_t x = 0; x < NX; x++) {
        for (size_t y = 0; y < NY; y++) {
            for (size_t f = 0; f < 9; ++f) {
                printf("%64.60f\n", var[x][y][f]);
            }
        }
    }
}


int main(int argc, char * const argv[])
{
    // Read arguments
    char* img_path = NULL;
    out_mode out = OUT_UNK;
    ssize_t max_iter = 0;
    
    while (optind < argc) {
        switch (getopt(argc, argv, "p:fi:")) {
            case 'p': { out = OUT_IMG; img_path = optarg; break; }
            case 'f': { out = OUT_FIN; break; }
            case 'i': { max_iter = strtol(optarg, NULL, 10); break; }
            default : { goto usage; }
        }
    }
    
    // check that execution mode is set (output images or fin values)
    if (out == OUT_UNK && max_iter < 1) {
    usage:
        fprintf(stderr, "usage: %s (-p <path> | -f) -i <iter> \n", basename((char*)argv[0]));
        fprintf(stderr, "  -p : output pictures in <path> directory\n");
        fprintf(stderr, "  -f : output populations values in stdout\n");
        fprintf(stderr, "  -i : Total number of iterations\n");
        return EXIT_FAILURE;
    }

    lbm_consts* h_consts = (lbm_consts*)malloc(sizeof(lbm_consts));
    
    initCol(h_consts->col[0],  1);
    initCol(h_consts->col[1],  0);
    initCol(h_consts->col[2], -1);
    initOpp(h_consts->opp);
    memcpy(h_consts->v, V, sizeof(V));
    memcpy(h_consts->t, T, sizeof(T));
    
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_consts), h_consts, sizeof(lbm_consts)));
        
    lbm_vars *h_vars = (lbm_vars*)malloc(sizeof(lbm_vars));
    initObstacles(h_vars);
    initVelocity(h_vars);
    initRho(h_vars);
    
    // Initialization of the populations at equilibrium with the given velocity.
    for (int y = 0; y < NY; y++) {
        for (int x = 0; x < NX; x++) {
            h_equilibrium(h_vars->fin[x][y], h_vars->rho[x][y], h_vars->vel[x][y]);
        }
    }
    
    lbm_vars *d_vars;
    HANDLE_ERROR(hipMalloc(&d_vars, sizeof(lbm_vars)));
    HANDLE_ERROR(hipMemcpy(d_vars, h_vars, sizeof(lbm_vars), hipMemcpyHostToDevice));

    pgm_image* pgm = pgm_create(NX, NY);

    for (int time = 0; time < max_iter; time++) {
        RUN_KERNEL_1D(lbm_right_wall,             NY, d_vars);
        RUN_KERNEL_2D(lbm_macro_and_left_wall, NX,NY, d_vars);
        RUN_KERNEL_1D(lbm_density,                NY, d_vars);
        RUN_KERNEL_2D(lbm_equilibrium_1,       NX,NY, d_vars);
        RUN_KERNEL_1D(lbm_equilibrium_2,          NY, d_vars);
        RUN_KERNEL_2D(lbm_collision,           NX,NY, d_vars);
        RUN_KERNEL_2D(lbm_streaming,           NX,NY, d_vars);

                // Visualization of the velocity.
        if (time % 100 == 0 && out == OUT_IMG) {
            HANDLE_ERROR(hipMemcpy(h_vars, d_vars, sizeof(lbm_vars), hipMemcpyDeviceToHost));

            for (size_t x = 0; x < NX; x++) {
                for (size_t y = 0; y < NY; y++) {
                    double vel = sqrt( SQUARE(h_vars->u[x][y][0]) + SQUARE(h_vars->u[x][y][1]) );
                    int color =  255 * vel * 10;
                    pgm_set_pixel(pgm, x, y, color);
                }
            }
            // build image file path and create it
            char* filename;
            asprintf(&filename, "%s/vel_%d.pgm", img_path, time/100);
            pgm_write(pgm, filename);
            free(filename);
        }
    }

    if (out == OUT_FIN) {
        print_variables(d_vars, h_vars, h_vars->fin);
    }

    pgm_destroy(pgm);
    free(h_consts);
    free(h_vars);
    HANDLE_ERROR(hipFree(d_vars));
    
    return EXIT_SUCCESS;
}
