#include "hip/hip_runtime.h"
/*!
 * \file    lbmFlowAroundCylinder.cu
 * \brief   Cuda version based on lbm_palabos_friendly (standard C).
 * \author  Adrien Python
 * \date    22.01.2017
 */

#include <stdlib.h>
#include <unistd.h>
#include <stdio.h>
#include <math.h>
#include <stdbool.h>
#include <libgen.h>
#include <pgm.h>

#define RE       220.0       // Reynolds number
#define NX       100//420         // Numer of lattice nodes (width)
#define NY       10 //180         // Numer of lattice nodes (height)
#define LY       ((NY) - 1)  // Height of the domain in lattice units
#define CX       ((NX) / 4)  // X coordinates of the cylinder
#define CY       ((NY) / 2)  // Y coordinates of the cylinder
#define R        ((NY) / 9)  // Cylinder radius
#define ULB      0.04        // Velocity in lattice units
#define NULB     ((ULB) * (R) / (RE))   // Viscoscity in lattice units
#define OMEGA    ((double)1. / (3*(NULB)+0.5))  // Relaxation parameter

#define SQUARE(a) ((a)*(a))
#define GPU_SQUARE(a) (__dmul_rn(a,a))

#define NB_THREADS 100

#define INDEX_2D_FROM_1D(x, y, i) (y) = (i)/(NX), (x) = (i)%(NX)

typedef enum { OUT_FIN, OUT_IMG, OUT_UNK } out_mode;

typedef struct {
    bool obstacles[NX][NY];  // Should reside in lbm_consts but is too big for constant memory
    double u[NX][NY][2];
    double feq[NX][NY][9];
    double fin[NX][NY][9];
    double fout[NX][NY][9];
    double rho[NX][NY];
    double vel[NX][NY][2];
} lbm_vars;

typedef struct {
    size_t col[3][3];
    size_t opp[9];
    ssize_t v[2][9];
    double t[9];
} lbm_consts;

__constant__ lbm_consts d_consts;

const ssize_t V[][9] = {
    { 1, 1, 1, 0, 0, 0,-1,-1,-1 },
    { 1, 0,-1, 1, 0,-1, 1, 0,-1 }
};
const double T[] = { 1./36, 1./9, 1./36, 1./9, 4./9, 1./9, 1./36, 1./9, 1./36 };

#define HANDLE_ERROR(ans) (handleError((ans), __FILE__, __LINE__))
inline void handleError(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(EXIT_FAILURE);
   }
}

#define HANDLE_KERNEL_ERROR(...) \
do {                                         \
    __VA_ARGS__;                             \
    HANDLE_ERROR( hipPeekAtLastError() );   \
    HANDLE_ERROR( hipDeviceSynchronize() ); \
} while(0)

/**
 * Setup: cylindrical obstacle and velocity inlet with perturbation
 * Creation of a mask with boolean values, defining the shape of the obstacle.
 */
static void initObstacles(lbm_vars* vars)
{
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            vars->obstacles[x][y] = SQUARE(x-CX) + SQUARE(y-CY) < SQUARE(R);
        }
    }
}

/**
 * Initial velocity profile: almost zero, with a slight perturbation to trigger
 * the instability.
 */
static void initVelocity(lbm_vars* vars)
{
    for (int d = 0; d < 2; d++) {
        for (int x = 0; x < NX; x++) {
            for (int y = 0; y < NY; y++) {
                vars->vel[x][y][d] = (1-d) * ULB * (1 + 0.0001 * sin( y / (double)LY * 2 * M_PI) );
            }
        }
    }
}

static void initRho(lbm_vars* vars)
{
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            vars->rho[x][y] = 1.0;
        }
    }
}

static void initCol(size_t* col, ssize_t v0)
{
    for (int f = 0, i = 0; f < 9 && i < 3; f++) {
        if (V[0][f] == v0) {
            col[i++] = f;
        }
    }
}

static void initOpp(size_t* opp)
{
    for (int f = 0; f < 9; f++) {
        for (int g = 0; g < 9; g++) {
            if (V[0][f] == -V[0][g] && V[1][f] == -V[1][g]) {
                opp[f] = g;
                break;
            }
        }
    }
}

__host__ static void h_equilibrium(double* feq, double rho, double* u)
{
    do {                                                                 
        double usqr = 3./2 * ( SQUARE(u[0]) + SQUARE(u[1]) );

        for (int f = 0; f < 9; f++) {
            double cu = 3 * ( V[0][f] * u[0] + V[1][f] * u[1] );
            feq[f] = rho * T[f] * ( 1 + cu + 0.5 * SQUARE(cu) - usqr );
        }                                                                
    } while(0);
}

__device__ static void d_equilibrium(double* feq, double rho, double* u)
{
    do {
        double usqr = __dmul_rn(3./2, __dadd_rn( GPU_SQUARE(u[0]), GPU_SQUARE(u[1]) ));
                                                                         
        for (int f = 0; f < 9; f++) {
            double cu = 3 * ( d_consts.v[0][f] * u[0] + d_consts.v[1][f] * u[1] );
            feq[f] = rho * d_consts.t[f] * ( 1 + cu + 0.5 * SQUARE(cu) - usqr );
        }                                                                
    } while(0);
}

__device__ static void macroscopic(double* fin, double* rho, double* u)
{   
    *rho = u[0] = u[1] = 0;

    for (int f = 0; f < 9; f++) {
        *rho += fin[f];

        u[0] += d_consts.v[0][f] * fin[f];
        u[1] += d_consts.v[1][f] * fin[f];
    }
    
    u[0] /= *rho;
    u[1] /= *rho;
}

__global__ void lbm_right_wall(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);
        
        if ( x == 0) {
            // Right wall: outflow condition.
            for (int i = 0; i < 3; i++) {
                int f = d_consts.col[2][i];
                d_vars->fin[NX-1][y][f] = d_vars->fin[NX-2][y][f];
            }
        }
    }
}

__global__ void lbm_macro_and_left_wall(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);

        // Compute macroscopic variables, density and velocity
        macroscopic(d_vars->fin[x][y], &d_vars->rho[x][y], d_vars->u[x][y]);
        
        // Left wall: inflow condition
        if (x == 0) {
            for (size_t d = 0; d < 2; d++) {
                d_vars->u[0][y][d] = d_vars->vel[0][y][d]; // TODO: collision!
            }   
        }
    }  
 }

__global__ void lbm_density(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);
   
        if (x == 0) {
            // Calculate the density
            double s2 = 0, s3 = 0;
            for (size_t i = 0; i < 3; i++) {
                s2 += d_vars->fin[0][y][d_consts.col[1][i]];
                s3 += d_vars->fin[0][y][d_consts.col[2][i]];
            }
            d_vars->rho[0][y] = 1./(1 - d_vars->u[0][y][0]) * (s2 + 2*s3);
        }
    }
}

__global__ void lbm_equilibrium_1(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);
   
        // Compute equilibrium
        d_equilibrium(d_vars->feq[x][y], d_vars->rho[x][y], d_vars->u[x][y]);
    }
}

__global__ void lbm_equilibrium_2(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);
   
        if (x == 0) {
            for (size_t i = 0, f = d_consts.col[0][i]; i < 3; f = d_consts.col[0][++i]) {
                d_vars->fin[0][y][f] = d_vars->feq[0][y][f] + d_vars->fin[0][y][d_consts.opp[f]] - d_vars->feq[0][y][d_consts.opp[f]];
            }
        }
    }
}

__global__ void lbm_collision(lbm_vars *d_vars)
{
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);
   
        for (size_t f = 0; f < 9; f++) {
            if (d_vars->obstacles[x][y]) {
                // Bounce-back condition for obstacle
                d_vars->fout[x][y][f] = d_vars->fin[x][y][d_consts.opp[f]];
            } else {
                // Collision step
                d_vars->fout[x][y][f] = __dadd_rn(__dmul_rn(-OMEGA, __dadd_rn(d_vars->fin[x][y][f], - d_vars->feq[x][y][f])), d_vars->fin[x][y][f]);
            }
        }
    }
}

__global__ void lbm_streaming(lbm_vars *d_vars)
{

    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < NX*NY; i += blockDim.x * gridDim.x) {
        int x, y;
        INDEX_2D_FROM_1D(x, y, i);

        // Streaming step
        for (size_t f = 0; f < 9; f++) {
            size_t x_dst = (x + NX + d_consts.v[0][f]) % NX;
            size_t y_dst = (y + NY + d_consts.v[1][f]) % NY;
            d_vars->fin[x_dst][y_dst][f] = d_vars->fout[x][y][f];
        }
   
    }
}

void print_variables(lbm_vars *d_vars, lbm_vars *h_vars, double var[NX][NY][9]) {

    HANDLE_ERROR(hipMemcpy(h_vars, d_vars, sizeof(lbm_vars), hipMemcpyDeviceToHost));

    for (size_t x = 0; x < NX; x++) {
        for (size_t y = 0; y < NY; y++) {
            for (size_t f = 0; f < 9; ++f) {
                printf("%64.60f\n", var[x][y][f]);
            }
        }
    }
}

int getThreads(int width, int height) {
    int dev, threads;
    hipDeviceProp_t prop;
    HANDLE_ERROR( hipGetDevice(&dev) );
    HANDLE_ERROR( hipGetDeviceProperties(&prop, dev) );

    int maxThreads = min (prop.maxThreadsDim[0], prop.maxThreadsPerBlock);
#ifdef NB_THREADS
    threads = NB_THREADS;
#else
    threads = prop.maxThreadsDim[0];
#endif

    if (threads > maxThreads)
        threads = maxThreads;

    return min(threads, width*height);
}

int main(int argc, char * const argv[])
{

    // Read arguments
    char* img_path = NULL;
    out_mode out = OUT_UNK;
    ssize_t max_iter = 0;
    
    while (optind < argc) {
        switch (getopt(argc, argv, "p:fi:")) {
            case 'p': { out = OUT_IMG; img_path = optarg; break; }
            case 'f': { out = OUT_FIN; break; }
            case 'i': { max_iter = strtol(optarg, NULL, 10); break; }
            default : { goto usage; }
        }
    }
    
    // check that execution mode is set (output images or fin values)
    if (out == OUT_UNK && max_iter < 1) {
    usage:
        fprintf(stderr, "usage: %s (-p <path> | -f) -i <iter> \n", basename((char*)argv[0]));
        fprintf(stderr, "  -p : output pictures in <path> directory\n");
        fprintf(stderr, "  -f : output populations values in stdout\n");
        fprintf(stderr, "  -i : Total number of iterations\n");
        return EXIT_FAILURE;
    }

    lbm_consts* h_consts = (lbm_consts*)malloc(sizeof(lbm_consts));
    
    initCol(h_consts->col[0],  1);
    initCol(h_consts->col[1],  0);
    initCol(h_consts->col[2], -1);
    initOpp(h_consts->opp);
    memcpy(h_consts->v, V, sizeof(V));
    memcpy(h_consts->t, T, sizeof(T));
    
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_consts), h_consts, sizeof(lbm_consts)));
        
    lbm_vars *h_vars = (lbm_vars*)malloc(sizeof(lbm_vars));
    initObstacles(h_vars);
    initVelocity(h_vars);
    initRho(h_vars);
    
    // Initialization of the populations at equilibrium with the given velocity.
    for (int y = 0; y < NY; y++) {
        for (int x = 0; x < NX; x++) {
            h_equilibrium(h_vars->fin[x][y], h_vars->rho[x][y], h_vars->vel[x][y]);
        }
    }
    
    lbm_vars *d_vars;
    HANDLE_ERROR(hipMalloc(&d_vars, sizeof(lbm_vars)));
    HANDLE_ERROR(hipMemcpy(d_vars, h_vars, sizeof(lbm_vars), hipMemcpyHostToDevice));

    dim3 dimBlock(1);
    dim3 dimGrid(getThreads(NX, NY));

    pgm_image* pgm = pgm_create(NX, NY);

    for (int time = 0; time < max_iter; time++) {
        HANDLE_KERNEL_ERROR(lbm_right_wall         <<<dimBlock, dimGrid>>>(d_vars));        
        HANDLE_KERNEL_ERROR(lbm_macro_and_left_wall<<<dimBlock, dimGrid>>>(d_vars));
        HANDLE_KERNEL_ERROR(lbm_density            <<<dimBlock, dimGrid>>>(d_vars));
        HANDLE_KERNEL_ERROR(lbm_equilibrium_1      <<<dimBlock, dimGrid>>>(d_vars));
        HANDLE_KERNEL_ERROR(lbm_equilibrium_2      <<<dimBlock, dimGrid>>>(d_vars));
        HANDLE_KERNEL_ERROR(lbm_collision          <<<dimBlock, dimGrid>>>(d_vars));
        HANDLE_KERNEL_ERROR(lbm_streaming          <<<dimBlock, dimGrid>>>(d_vars));

        // Visualization of the velocity.
        if (time % 100 == 0 && out == OUT_IMG) {
            HANDLE_ERROR(hipMemcpy(h_vars, d_vars, sizeof(lbm_vars), hipMemcpyDeviceToHost));

            double vel[NX][NY];
            for (size_t x = 0; x < NX; x++) {
                for (size_t y = 0; y < NY; y++) {
                    vel[x][y] = sqrt( SQUARE(h_vars->u[x][y][0]) + SQUARE(h_vars->u[x][y][1]) );
                    int color =  255 * vel[x][y] * 10;
                    pgm_set_pixel(pgm, x, y, color);
                }
            }
            // build image file path and create it
            char* filename;
            asprintf(&filename, "%s/vel_%d.pgm", img_path, time/100);
            pgm_write(pgm, filename);
            free(filename);
        }
    }

    if (out == OUT_FIN) {
        print_variables(d_vars, h_vars, h_vars->fin);
    }

    pgm_destroy(pgm);
    free(h_consts);
    free(h_vars);
    HANDLE_ERROR(hipFree(d_vars));
    
    return EXIT_SUCCESS;
}
